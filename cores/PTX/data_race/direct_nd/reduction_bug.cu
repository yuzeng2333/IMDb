
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
using namespace std;
        /* a sum reduction on the array of floats 'in'.
         * The reduction result is written to the
          * address 'result'.  The number of elements to
           * be reduced is given by 'size'
            *
             * The example contains data races because barrier
              * synchronisation statements, of the form:
               *   __syncthreads();
                * are missing.
                 *
                  * Can you add them to eliminate all data races?
                   */

#define N 8 /* Same as blockDim */

#define tid threadIdx.x

__global__ void reduce(int *in, int *result, int size) {

      __shared__ int partial_sums[N];

        /* Each thread sums elements
             in[tid], in[tid + N], in[tid + 2*N], ...
               */
        partial_sums[tid] = in[tid];
        for(int i = tid + N; i < size; i += N) {
                 partial_sums[i] += in[i];
                  }

                               /* Tree reduction computes final sum into partial_sums[0] */
                                 for(int d = N/2; d > 0; d >>= 1) {
                                         if(tid < d) {
                                                   partial_sums[tid] += partial_sums[tid + d];
                                                       }
                                                         }

                                                           /* Master thread writes out result */
                                                             if(tid == 0) {
                                                                     *result = partial_sums[0];
                                                                       }
                                                                         
}

